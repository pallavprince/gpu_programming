
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define  my_assert(expression)	\
	if (expression)		\
		asm("trap;")

#define gpuErrchk(ans)					\
	{						\
		gpuAssert((ans), __FILE__, __LINE__);	\
	}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void my_divergent_kernel(int *x, int *y, int *z)
{
	my_assert((*x == *y || *x == *z));
	if (*x == *y)
		*z = *x;
	else
		*z = *y;
}

__global__ void my_non_divergent_kernel(int *x, int *y, int *z)
{
	int predicate;

	my_assert((*x == *y || *x == *z));
	predicate = (*x == *y);
	if (predicate)
		*z = *x;
	if (!predicate)
		*z = *y;
}

static int process_args(int argc, char *argv[])
{
	int nth;

	if (argc != 2) {
		printf("%s <number of threads>\n", argv[0]);
		return 0;
	}
	nth = atoi(argv[1]);
	if (nth > 1024) {
		printf("More than 1024 threads are not supported\n");
		return 0;
	}
	return nth;
}

int main(int argc, char *argv[])
{
	int hx = 3, hy = 4, hz = 5;
	hipEvent_t start,stop;
	int *dx, *dy, *dz, nth;
	dim3 grid(1,1,1);
	int ox, oy, oz;
	float ms;

	nth = process_args(argc, argv);
	if (!nth)
		exit(-1);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc(&dx, sizeof(int));
	hipMalloc(&dy, sizeof(int));
	hipMalloc(&dz, sizeof(int));

	printf("Original values: x = %d, y = %d, z = %d\n", hx, hy, hz);
	hipMemcpy(dx, &hx, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dy, &hy, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dz, &hz, sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(start);
	my_divergent_kernel<<<grid, nth>>>(dx, dy, dz);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms,start,stop);
	gpuErrchk(hipPeekAtLastError());

	/* cudaDeviceSynchronize(); not needed due to implicit barrier */
	hipMemcpy(&ox, dx, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&oy, dy, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&oz, dz, sizeof(int), hipMemcpyDeviceToHost);
	printf("Exchanged values using divergent kernel: x = %d, y = %d, z = %d\n",
		ox, oy, oz);
	printf("divergent kernel runtime = %f msec\n\n", ms);


	printf("Original values: x = %d, y = %d, z = %d\n", hx, hy, hz);
	hipMemcpy(dx, &hx, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dy, &hy, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dz, &hz, sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(start);
	my_non_divergent_kernel<<<grid, nth>>>(dx, dy, dz);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms,start,stop);
	gpuErrchk(hipPeekAtLastError());

	/*cudaDeviceSynchronize(); not needed due to implicit barrier */
	hipMemcpy(&ox, dx, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&oy, dy, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&oz, dz, sizeof(int), hipMemcpyDeviceToHost);
	printf("Exchanged values using non-divergent kernel: x = %d, y = %d, z = %d\n",
		ox, oy, oz);
	printf("non-divergent kernel runtime = %f msec\n", ms);

	hipEventDestroy(stop);
	hipEventDestroy(start);
	hipFree(dx);
	hipFree(dy);
	hipFree(dz);
	return 0;
}
